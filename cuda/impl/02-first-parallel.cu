
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */
__global__
void firstParallel()
{
  // printf("This should be running in parallel.\n");
  printf("Yes, i'm running in parallel <%d, %d>\n", blockIdx.x, threadIdx.x);
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */

  firstParallel<<<3, 3>>>();
  hipDeviceSynchronize();

  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */

}
