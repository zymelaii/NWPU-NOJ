
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  } else {
    // print too much, we do not need that!
    // printf("Failure. Update the execution configuration as necessary.\n");
  }
}

int main()
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */

  // printSuccessForCorrectExecutionConfiguration<<<1, 1>>>();
  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();
  hipDeviceSynchronize();
}
