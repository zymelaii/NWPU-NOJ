#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include <stdio.h>
#include <hiprand.h>
#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )
#define SIZE 4096
#define BLKDIM 32
// A single-threaded version of matrix multiply
void host_sgemm( int m, int n, int k, float *a, float *b, float *c ){
  for( int j = 0; j < n; j++ )  {
    for( int i = 0; i < m; i++ )    {
      for( int koff = 0; koff < k; koff++ )      {
        c[INDX(i, j, m)] += a[INDX( i, koff, m )] * b[INDX( koff, j, n )];
      } /* end for i */
    } /* end jb */
  } /* end for j */
} /* end host_sgemm */



  /* Compute r = p * q, for square nxn matrices p, q, r; this version
  does not use shared memory. This kernel does not require that n is
  a multiple of BLKDIM */
__global__ void matmul(const float *p, const float *q, float *r, int n)
{
	const int i = blockIdx.y * blockDim.y + threadIdx.y;
	const int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k;
	float val = 0.0;
	if (i < n && j < n) {
		for (k = 0; k<n; k++) {
			val += p[i*n + k] * q[k*n + j];
		}
		r[i*n + j] = val;
	}
}

/* Same as above, but does not require that n is a multiple of
BLKDIM. To do so, it fills shared buffers so that values outside
the matrices are treated as zeros. */
__global__ void matmulb_generic(const float *p, const float *q, float *r, int n)
{
	__shared__ float local_p[BLKDIM][BLKDIM];
	__shared__ float local_q[BLKDIM][BLKDIM];
	const int bx = blockIdx.x, by = blockIdx.y;
	const int tx = threadIdx.x, ty = threadIdx.y;
	const int i = by * BLKDIM + ty;
	const int j = bx * BLKDIM + tx;
	float v = 0.0; int m, k;
	for (m = 0; m < n; m += BLKDIM) { /* loop over tiles */
		local_p[ty][tx] = local_q[ty][tx] = 0;
		if (i<n && m + tx<n)
			local_p[ty][tx] = p[i*n + (m + tx)];
		if (j<n && m + ty<n)
			local_q[ty][tx] = q[(m + ty)*n + j];

		__syncthreads();

		for (k = 0; k < BLKDIM; k++) { /* loop within tile */
			v += local_p[ty][k] * local_q[k][tx];
		}

		__syncthreads();
	}
	if (i<n && j<n)
		r[i*n + j] = v; /* write result to global memory */
}



int main()
{
    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    float *a, *b, *c, *cdef;

    size_t numbytes = size * size * sizeof( float );

    hipEvent_t start, stop; hipEventCreate(&start);	hipEventCreate(&stop); float runtime;
	clock_t start_t, finish_t;

    // Allocate all our host-side (CPU) and device-side (GPU) data
    hipMallocManaged( (void **)&a, numbytes);
    hipMallocManaged( (void **)&b, numbytes);
    hipMallocManaged( (void **)&c, numbytes);
    hipMallocManaged( (void **)&cdef, numbytes);

    if( a == NULL || b == NULL || c == NULL || c == NULL)
    {
      fprintf(stderr,"Error in malloc\n");
      return 911;
    }

    // Clear the result matrices to zero
    memset( c, 0, numbytes );    memset( cdef, 0, numbytes );




	start_t = clock();
    // Initialize the a and b matrices to random data
    for( int i = 0; i < size * size; i++ )
    {
      a[i] = float( rand() ) / ( float(RAND_MAX) + 1.0 );
      b[i] = float( rand() ) / ( float(RAND_MAX) + 1.0 );
    }

	finish_t = clock();
	runtime = (float)(finish_t - start_t) / CLOCKS_PER_SEC * 1000;
	fprintf(stdout, "Initialize random data in CPU: %f ms\n", runtime);


	// First run the CPU verison of dgemm so we can compare the results
	//start_t = clock();
	//printf("Launching CPU sgemm\n");
	//host_sgemm(size, size, size, a, b, cdef);
	//finish_t = clock();
	//runtime = (float)(finish_t - start_t) / CLOCKS_PER_SEC * 1000;
	//fprintf(stdout, "Total time CPU is %f ms\n", runtime);
	//fprintf(stdout, "Performance is %f GFlop/s\n",
	//	2.0 * (double)size * (double)size * (double)size /
	//	((double)runtime / 1000.0) * 1.e-9);


	start_t = clock();
	// Generate size * size random numbers
	printf("Create random numbers\n");
	// Create pseudo-random number generator
	hiprandGenerator_t gen;
	hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateNormal(gen, a, size*size, 0.0, float(RAND_MAX));
	hiprandGenerateNormal(gen, b, size*size, 0.0, float(RAND_MAX));
	finish_t = clock();
	runtime = (float)(finish_t - start_t) / CLOCKS_PER_SEC * 1000;
	fprintf(stdout, "Initialize via cuRAND on CPU: %f ms\n", runtime);


  start_t = clock();
	// Generate size * size random numbers
	printf("Create random numbers\n");
	// Create pseudo-random number generator
	hiprandGenerator_t gen1;
	hiprandCreateGenerator(&gen1, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateNormal(gen1, a, size*size, 0.0, float(RAND_MAX));
	hiprandGenerateNormal(gen1, b, size*size, 0.0, float(RAND_MAX));
	finish_t = clock();
	runtime = (float)(finish_t - start_t) / CLOCKS_PER_SEC * 1000;
  fprintf(stdout, "Initialize via cuRAND on GPU: %f ms\n", runtime);






	/**
	** Matrix-matrix multiply WITHOUT shared memory
	**/
	dim3 block(BLKDIM, BLKDIM);
	dim3 grid((SIZE + BLKDIM - 1) / BLKDIM, (SIZE + BLKDIM - 1) / BLKDIM);
	printf("No shared memory:\t");
	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start, 0);
	matmul << <grid, block >> >(a, b, c, SIZE);	hipDeviceSynchronize();
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&runtime, start, stop);
	printf("GPU Elapsed time:%.6f ms.\n", runtime);
	fprintf(stdout, "Performance is %f GFlop/s\n",
		2.0 * (double)size * (double)size * (double)size /
		((double)runtime / 1000.0) * 1.e-9);

	/**
	** Matrix-matrix multiply WITH shared memory
	**/
	printf("Shared memory:\t");
	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start, 0);
	matmulb_generic << <grid, block >> >(a, b, c, SIZE);    hipDeviceSynchronize();
	hipEventRecord(stop, 0);	hipEventSynchronize(stop);	hipEventElapsedTime(&runtime, start, stop);
	printf("GPU Elapsed time:%.6f ms.\n", runtime);
	fprintf(stdout, "Performance is %f GFlop/s\n",
		2.0 * (double)size * (double)size * (double)size /
		((double)runtime / 1000.0) * 1.e-9);

    // Now run the GPU version of sgemm using the cuBLAS library
    hipblasHandle_t handle;   hipblasStatus_t stat = hipblasCreate( &handle );

    // Set these constants so we get a simple matrix multiply with hipblasDgemm
    float alpha = 1.0;    float beta  = 0.0;

    // Launch hipblasSgemm on the GPU
    printf("Launching GPU sgemm\n");    hipEventRecord(start, 0);
    hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha,
                 a, size,
                 b, size,
                 &beta,
                 c, size );
    hipEventRecord(stop, 0);	hipEventSynchronize(stop);
    hipEventElapsedTime(&runtime, start, stop);


    fprintf(stdout, "Total time GPU CUBLAS is %f ms\n", runtime );
    fprintf(stdout, "Performance is %f GFlop/s\n",
      2.0 * (double) size * (double) size * (double) size /
      ( (double) runtime / 1000.0 ) * 1.e-9 );

    hipblasDestroy( handle );
	hiprandDestroyGenerator(gen);hiprandDestroyGenerator(gen1);

    // Do some error checking to verify our GPU & CPU verisons are within
    // an acceptable error bound
    float temp = 0.0;
    for( int i = 0; i < size * size; i++ )
    {
        temp += ( c[i] - cdef[i] ) * ( c[i] - cdef[i] );
    } /* end for */

    printf("error is %f\n",temp);
    if( temp > 10 ) printf("Error value is suspiciously high!\n");

    hipFree( a );
    hipFree( b );
    hipFree( c );
    hipFree( cdef );

    return 0;
}
