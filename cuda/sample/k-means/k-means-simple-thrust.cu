#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

// In the assignment step, each point (thread) computes its distance to each
// cluster centroid and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(const thrust::device_ptr<float> data_x,
                                const thrust::device_ptr<float> data_y,
                                int data_size,
                                const thrust::device_ptr<float> means_x,
                                const thrust::device_ptr<float> means_y,
                                thrust::device_ptr<float> new_sums_x,
                                thrust::device_ptr<float> new_sums_y,
                                int k,
                                thrust::device_ptr<int> counts) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  atomicAdd(thrust::raw_pointer_cast(new_sums_x + best_cluster), x);
  atomicAdd(thrust::raw_pointer_cast(new_sums_y + best_cluster), y);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(thrust::device_ptr<float> means_x,
                                  thrust::device_ptr<float> means_y,
                                  const thrust::device_ptr<float> new_sum_x,
                                  const thrust::device_ptr<float> new_sum_y,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

int main(int argc, const char* argv[]) {
  if (argc < 3) {
    std::cerr << "usage: assign_clusters <data-file> <k> [iterations]"
              << std::endl;
    std::exit(EXIT_FAILURE);
  }

  const auto k = std::atoi(argv[2]);
  const auto number_of_iterations = (argc == 4) ? std::atoi(argv[3]) : 300;

  thrust::host_vector<float> h_x;
  thrust::host_vector<float> h_y;
  std::ifstream stream(argv[1]);
  std::string line;
  while (std::getline(stream, line)) {
    std::istringstream line_stream(line);
    float x, y;
    uint16_t label;
    line_stream >> x >> y >> label;
    h_x.push_back(x);
    h_y.push_back(y);
  }

  const size_t number_of_elements = h_x.size();

  thrust::device_vector<float> d_x = h_x;
  thrust::device_vector<float> d_y = h_y;

  std::mt19937 rng(std::random_device{}());
  std::shuffle(h_x.begin(), h_x.end(), rng);
  std::shuffle(h_y.begin(), h_y.end(), rng);
  thrust::device_vector<float> d_mean_x(h_x.begin(), h_x.begin() + k);
  thrust::device_vector<float> d_mean_y(h_y.begin(), h_y.begin() + k);

  thrust::device_vector<float> d_sums_x(k);
  thrust::device_vector<float> d_sums_y(k);
  thrust::device_vector<int> d_counts(k, 0);

  const int threads = 1024;
  const int blocks = (number_of_elements + threads - 1) / threads;

  const auto start = std::chrono::high_resolution_clock::now();
  for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
    thrust::fill(d_sums_x.begin(), d_sums_x.end(), 0);
    thrust::fill(d_sums_y.begin(), d_sums_y.end(), 0);
    thrust::fill(d_counts.begin(), d_counts.end(), 0);

    assign_clusters<<<blocks, threads>>>(d_x.data(),
                                         d_y.data(),
                                         number_of_elements,
                                         d_mean_x.data(),
                                         d_mean_y.data(),
                                         d_sums_x.data(),
                                         d_sums_y.data(),
                                         k,
                                         d_counts.data());
    hipDeviceSynchronize();

    compute_new_means<<<1, k>>>(d_mean_x.data(),
                                d_mean_y.data(),
                                d_sums_x.data(),
                                d_sums_y.data(),
                                d_counts.data());
    hipDeviceSynchronize();
  }
  const auto end = std::chrono::high_resolution_clock::now();
  const auto duration =
      std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  std::cerr << "Took: " << duration.count() << "s" << std::endl;

  thrust::host_vector<float> h_mean_x = d_mean_x;
  thrust::host_vector<float> h_mean_y = d_mean_y;

  for (size_t cluster = 0; cluster < k; ++cluster) {
    std::cout << h_mean_x[cluster] << " " << h_mean_y[cluster] << std::endl;
  }
}
